#include "hip/hip_runtime.h"
#include "./utilities/common.cuh"
#include "./utilities/nep_utilities.cuh"
#include <iostream>

template<typename DType>
__global__ void feat_2b_calc(
        const DType * coeff2,
        const DType * d12_radial,
        const int * NL_radial,
        const int * atom_map,
        const DType rcut_radial,
        const DType rcinv_radial,
        DType * feat_2b,
        DType * dfeat_c2,
        DType * dfeat_2b,
        const int batch_size,
        const int natoms,
        const int neigh_num,
        const int n_max,
        const int n_base,
        const int num_types,
        const int num_types_sq)
{  
    // 计算全局线程索引，每个线程处理一个中心原子
    int global_atom_index = blockIdx.x * blockDim.x + threadIdx.x;
    // 计算批次和原子索引
    int batch_id = global_atom_index / natoms;
    int atom_id = global_atom_index % natoms;
    int c_index = 0;
    if (batch_id < batch_size && atom_id < natoms) {
        int t1 = atom_map[atom_id];
        DType q[MAX_DIM] = {static_cast<DType>(0.0)};
        int neigh_start_idx = batch_id * natoms * neigh_num + atom_id * neigh_num;
        int r12_start_idx =  batch_id * natoms * neigh_num * 4 + atom_id * neigh_num * 4;
        int feat_start_idx = batch_id * natoms * n_max + atom_id * n_max; 
        int dfeat_c_start_idx = batch_id * natoms * num_types * n_base + atom_id * num_types * n_base;
        int dfeat_2b_start_idx = batch_id * natoms * neigh_num * n_max + atom_id * neigh_num * n_max;
        int c_start_idx = t1 * num_types * n_max * n_base;

        for (int i1=0; i1 < neigh_num; ++i1) {
            int n2 = NL_radial[neigh_start_idx + i1]-1;
            if (n2 < 0) return;
            int t2 = atom_map[n2];
            int c_I_J_idx = c_start_idx + t2 * n_max * n_base;
            int rij_idx = r12_start_idx + i1*4;
            int d2b_idx = dfeat_2b_start_idx + i1 * n_max;
            DType d12 = d12_radial[rij_idx]; // [rij, x, y, z]
            DType fc12, fcp12;
            find_fc_and_fcp(rcut_radial, rcinv_radial, d12, fc12, fcp12);
            DType fn12[MAX_NUM_N];
            DType fnp12[MAX_NUM_N];
            find_fn_and_fnp(
                n_base, rcinv_radial, d12, fc12, fcp12, fn12, fnp12);
            for (int n = 0; n < n_max; ++n) {
                DType gn12 = static_cast<DType>(0.0);
                for (int k = 0; k < n_base; ++k) {
                    // c2的维度为[Nmax, Nbas, I, J]对c的索引会更方便
                    c_index =  c_I_J_idx + n * n_base + k;
                    gn12 += fn12[k] * coeff2[c_index];
                    dfeat_2b[d2b_idx + n] += fnp12[k]*coeff2[c_index];
                    // if (n == 0 and k == 0) {
                    //     printf("batch %d I %d J %d n %d k %d c %f cid %d rij %f rid %d\n", batch_id, atom_id, n2, n, k, coeff2[c_index], c_index, d12, rij_idx);
                    // }
                    if (n == 0) {
                        dfeat_c2[dfeat_c_start_idx + t2 * n_base + k] += fn12[k]; //[batch, n_atom, J_Ntypes, N_base]
                    }
                }
                feat_2b[feat_start_idx + n] += gn12;
            }
        }//neighs
        // printf("batch %d atom %d feat [%f %f %f %f %f] dfc [%f %f]\n", batch_id, atom_id, 
        //     feat_2b[feat_start_idx], feat_2b[feat_start_idx+1], feat_2b[feat_start_idx+2], feat_2b[feat_start_idx+3], feat_2b[feat_start_idx+4],
        //         dfeat_c2[dfeat_c_start_idx], dfeat_c2[dfeat_c_start_idx+1*n_base]);
    }
}

template<typename DType>
void launch_calculate_nepfeat(
        const DType * coeff2,
        const DType * d12_radial,
        const int * NL_radial,
        const int * atom_map,
        const double rcut_radial,
        DType * feat_2b,
        DType * dfeat_c2,
        DType * dfeat_2b,
        const int batch_size,
        const int natoms,
        const int neigh_num,
        const int n_max,
        const int n_base,
        const int num_types,
        const int device_id
) {
    hipSetDevice(device_id);
    int num_types_sq = num_types * num_types;
    int BLOCK_SIZE = 64;
    int grid_size = (natoms * batch_size - 1) / BLOCK_SIZE + 1;
    // float rcinv_radial = 1/rcut_radial;
    DType rcinv_radial = static_cast<DType>(1.0 / rcut_radial);
    feat_2b_calc<<<grid_size, BLOCK_SIZE>>>(
                coeff2, d12_radial, NL_radial, atom_map, 
                    static_cast<DType>(rcut_radial), rcinv_radial,
                        feat_2b, dfeat_c2, dfeat_2b, 
                            batch_size, natoms, neigh_num, 
                                n_max, n_base, num_types, num_types_sq
                            );
                            
    // hipDeviceSynchronize();
    // 打印 dfeat_c2 数据 (将 dfeat_c2 从设备内存复制到主机内存)
    // DType * h_dfeat_c2 = new DType[batch_size * natoms * num_types * n_base];  // 主机内存中的副本
    // hipMemcpy(h_dfeat_c2, dfeat_c2, batch_size * natoms * num_types * n_base * sizeof(DType), hipMemcpyDeviceToHost);
    
    // // 打印 dfeat_c2 的一部分 (例如前 5 个元素)
    // printf("dfeat_c2 (first few values):\n");
    // for (int i = 0; i < batch_size; ++i) {
    //     for (int j = 0; j < natoms; ++j) {
    //         for (int k = 0; k < num_types; ++k) {
    //             for (int p = 0; p < n_base; ++p) {
    //                 printf("dfeat_c2[%d][%d][%d][%d] = %f\n", i, j, k, p, h_dfeat_c2[i * natoms * num_types * n_base + j* num_types * n_base + k * n_base + p]);
    //             }
    //         }
    //     }
    // }

    // 打印 dfeat_2b 数据 (将 dfeat_2b 从设备内存复制到主机内存)
    // DType * h_dfeat_2b = new DType[batch_size * natoms * neigh_num * n_max];  // 主机内存中的副本
    // hipMemcpy(h_dfeat_2b, dfeat_2b, batch_size * natoms * neigh_num * n_max * sizeof(DType), hipMemcpyDeviceToHost);
    
    // 打印 dfeat_2b 的一部分 (例如前 5 个元素)
    // printf("dfeat_2b (first few values):\n");
    // for (int i = 0; i < batch_size; ++i) {
    //     for (int j = 0; j < 10; ++j) {
    //         for (int k = 0; k < neigh_num; ++k) {
    //             printf("dfeat_2b[%d][%d][%d] = ", i, j, k);
    //             for (int l = 0; l < n_max; ++l) {
    //                 printf(" %f ", h_dfeat_2b[i * natoms * neigh_num * n_max + j * neigh_num * n_max + k * n_max + l]);
    //             }
    //             printf("\n");
    //         }
    //     }
    // }
}

template void launch_calculate_nepfeat(
            const float * coeff2,
            const float * d12_radial,
            const int * NL_radial,
            const int * atom_map,
            const double rcut_radial,
            float * feat_2b,
            float * dfeat_c2,
            float * dfeat_2b,
            const int batch_size,
            const int natoms,
            const int neigh_num,
            const int n_max,
            const int n_base,
            const int num_types,
            const int device_id
                );

template void launch_calculate_nepfeat(
            const double * coeff2,
            const double * d12_radial,
            const int * NL_radial,
            const int * atom_map,
            const double rcut_radial,
            double * feat_2b,
            double * dfeat_c2,
            double * dfeat_2b,
            const int batch_size,
            const int natoms,
            const int neigh_num,
            const int n_max,
            const int n_base,
            const int num_types,
            const int device_id
    );