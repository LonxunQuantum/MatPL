
#include <hip/hip_runtime.h>
#include <iostream>

template<typename DType>
__global__ void compress_calc_grad(
    DType * Grad,
    const DType * f2,
    const DType * coefficient,
    const DType * grad_output,
    const int sij_num,
    const int layer_node,
    const int coe_num
    )
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < sij_num) {
        DType f2_val = f2[tid];
        int coefficient_base = tid * layer_node * coe_num;

        for (int i = 0; i < layer_node; i++) {
            DType coeff_0 = coefficient[coefficient_base + i * coe_num];
            DType coeff_1 = coefficient[coefficient_base + i * coe_num + 1];
            DType coeff_2 = coefficient[coefficient_base + i * coe_num + 2];
            // DType coeff_3 = coefficient[tid * layer_node * 4 + i * 4 + 3];
            if (coe_num == 4) {
            Grad[tid * layer_node + i] = 3.0f * f2_val * f2_val * coeff_0 + 2.0f * f2_val * coeff_1 + coeff_2;
            } else {
                DType coeff_3 = coefficient[coefficient_base + i * coe_num + 3];
                DType coeff_4 = coefficient[coefficient_base + i * coe_num + 4];
                Grad[tid * layer_node + i] = 5.0f * f2_val * f2_val * f2_val * f2_val * coeff_0 +
                                                4.0f * f2_val * f2_val * f2_val * coeff_1 +
                                                3.0f * f2_val * f2_val * coeff_2 +
                                                2.0f * f2_val * coeff_3 +
                                                coeff_4;
            }
            // Grad[tid * layer_node] = Grad[tid * layer_node]  + (3 * f2_val * f2_val * coeff_0 + 2 * f2_val * coeff_1 + coeff_2) * grad_output[tid * layer_node + i];
        }
    }
}

template<typename DType>
void launch_calculate_compress_grad(
    const DType * f2,
    const DType * coefficient,
    const DType * grad_output,
    const int sij_num,
    const int layer_node,
    const int coe_num,
    DType * Grad
) {
    const int blockSize = 256;
    const int gridSize = (sij_num + blockSize -1) / blockSize;
    // Launch the kernel
    compress_calc_grad<<<gridSize, blockSize>>>(Grad, f2, coefficient, grad_output, sij_num, layer_node, coe_num);
}

template void launch_calculate_compress_grad(
    const float * f2,
    const float * coefficient,
    const float * grad_output,
    const int sij_num,
    const int layer_node,
    const int coe_num,
    float * Grad
    );

template void launch_calculate_compress_grad(
    const double * f2,
    const double * coefficient,
    const double * grad_output,
    const int sij_num,
    const int layer_node,
    const int coe_num,
    double * Grad
    );